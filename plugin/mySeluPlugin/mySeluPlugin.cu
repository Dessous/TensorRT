#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "mySeluPlugin.h"
#include <hip/hip_fp16.h>


    __global__ void kernelSelu(
        int N,
        int iH,
        int iW,
        float alpha,
        float lambda,
        float* outputs
        )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N){
        outputs[index] = outputs[index] >= 0 ? outputs[index] * lambda : alpha * lambda * (expf(outputs[index]) - 1);
    }
    __syncthreads();
}

int inferenceSelu(
    int batchSize,
    int iC,
    int iH,
    int iW,
    float alpha,
    float lambda,
    float* inputs,
    float* outputs,
    hipStream_t stream){
        // NCHW
        const int nThreads = 512;
        int len = iC * iH * iW;

        int nBlocks = (int)((float)len / nThreads) + 1;

        for(int i=0; i<batchSize; ++i){
            // NOTE: kernelCopy kernel can be replaced with hipMemcpy function
            hipMemcpy(outputs, inputs, sizeof(float) * len, hipMemcpyDeviceToDevice);
            kernelSelu<<<nBlocks, nThreads, 0, stream>>>(len, iH, iW, alpha, lambda, outputs);
            outputs += len;
            inputs += len;
        }

    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                __FILE__, __LINE__, hipGetErrorString( err ) );
        return 1;
    }
    return 0;
}

int mySeluPlugin::enqueue(
    int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
{
    return inferenceSelu(batchSize, iC, iH, iW, alpha, lambda, (float*)inputs[0], (float*)outputs[0], stream);
}
